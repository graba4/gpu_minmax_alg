#include "hip/hip_runtime.h"
#include <string.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "resources.h"
#include <stdlib.h>

__global__ void print_ref(double *ref, int len);
cuda_matrix* allocate_recources(io_info *info);
void free_matrix(double *matrix);
void free_matrix(cuda_matrix *matrix);
void create_matrix(cuda_matrix *matrix, int arrlen, bool clear, unsigned int seed);
void print_matrix(double *matrix, int length);
void gen_reference(cuda_matrix *matrix, double *h_matrix, int length);
void print_dev_info();

cuda_matrix* allocate_recources(io_info *info)
{
	hipError_t error;
	int arrlen = info->v_opt;

	cuda_matrix *matrix = (cuda_matrix*)malloc(sizeof(cuda_matrix));
	assert(matrix != NULL);
	matrix->arrlen = arrlen;
	matrix->core_count = info->c_opt;
	matrix->thread_count = info->t_opt;
	matrix->window_size = info->w_opt;

	create_matrix(matrix, arrlen, false, info->seed);
	info->seed = matrix->seed;

	error = hipMalloc(&(matrix->d_solution), sizeof(double)*arrlen);
	checkCudaErrors(error);

	return matrix;
}

#define DEC_FACTOR (1000)
void create_matrix(cuda_matrix *matrix, int arrlen, bool clear, unsigned int seed)
{
	hipError_t error;
	double *h_matrix;

	if (clear) {
		h_matrix = (double *)calloc(arrlen, sizeof(double));
		assert(h_matrix != NULL);
	} else {
		h_matrix = (double *)malloc(sizeof(double) * arrlen);
		assert(h_matrix != NULL);
		seed = (seed==0)? time(NULL) : seed;
		matrix->seed = seed;

		for (int i = 0; i < arrlen; ++i){
			srand(seed*(i+1)*(arrlen+1)+i);
			h_matrix[i] = ((double)rand()/(double)RAND_MAX)*DEC_FACTOR;
			assert(h_matrix[i] > 0);
		}
	}

	//print_matrix(h_matrix, arrlen);

	error = hipMalloc(&(matrix->d_matrix), sizeof(double)*arrlen);
	checkCudaErrors(error);

	error = hipMalloc(&(matrix->d_minval), sizeof(double)*arrlen);
	checkCudaErrors(error);

	error = hipMalloc(&(matrix->d_maxval), sizeof(double)*arrlen);
	checkCudaErrors(error);

	error = hipMemcpy(matrix->d_matrix, h_matrix, arrlen*sizeof(double), hipMemcpyHostToDevice);
	checkCudaErrors(error);


	free_matrix(h_matrix);
}

void free_matrix(cuda_matrix *matrix)
{
	assert(matrix != NULL);
	hipError_t error;
	//error = hipFree(matrix->d_reference);
	//checkCudaErrors(error);
	error = hipFree(matrix->d_matrix);
	checkCudaErrors(error);
	error = hipFree(matrix->d_maxval);
	checkCudaErrors(error);
	error = hipFree(matrix->d_minval);
	checkCudaErrors(error);
	//error = hipFree(matrix->d_solution);
	//checkCudaErrors(error);

	free(matrix);
}

//not used
void gen_reference(cuda_matrix *matrix, double *h_matrix, int length)
{
	hipError_t error;

	error = hipMalloc(&(matrix->d_reference), sizeof(double)*(length+1));
	checkCudaErrors(error);
	error = hipMemcpy(matrix->d_reference, h_matrix, sizeof(double)*(length+1), hipMemcpyHostToDevice);
	checkCudaErrors(error);
}

void free_matrix(double *matrix)
{
	assert(matrix != NULL);
	free(matrix);
}

void print_matrix(double *matrix, int length)
{
	assert(matrix != NULL);

	/* image row */
	for (int i = 0; i < length; i++){
		printf("%.1f ", (matrix[i] == -0.0)? 0.0 : matrix[i]);
	}
	printf("\n");
}

//not used
__global__ void print_ref(double *ref, int len)
{
	for (int i = 0; i < len+1; ++i)
	{
		printf("%.1f ", ref[i]);
	}
	printf("\n");
}

void print_dev_info()
{
	hipDeviceProp_t prop;

	int count;
	checkCudaErrors(hipGetDevice(&count));
	for (int i = 0; i < 1; ++i)
	{
		checkCudaErrors(hipGetDeviceProperties(&prop, i));

		printf("----CUDA-DEVICE----\n");
		printf("Name: %s\n", prop.name);
		printf("Clock Rate: %d\n", prop.clockRate);
		printf("Memory: %zuMB\n", prop.totalGlobalMem/1000000);
		printf("Multiprocessors: %d\n", prop.multiProcessorCount);
		printf("Threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Shared Mem per block: %zu\n", prop.sharedMemPerBlock);
		//printf("Gridsize x: %d y: %d\n", prop.maxGridSize[0], prop.maxGridSize[1]);

		printf("\n");
	}
}
