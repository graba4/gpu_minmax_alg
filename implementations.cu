#include "hip/hip_runtime.h"
#include "implementations.h"
#include "io.h"
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "resources.h"
#include <timer.h>
#include "errors.h"
#include "cuda_schedule.h"

#define ROUND_UP(N, S) (N%S == 0) ? N/S : N/S+1
#define BETWEEN(value, min, max) (value < max && value > min)
#define DEV_ID (0)

__global__ void naive_aproach_one_thread(cuda_matrix *matrix);

double naive_aproach_fabian(cuda_matrix *matrix){
	hipDeviceProp_t prop;
	checkCudaErrors(hipGetDeviceProperties(&prop, DEV_ID));

	int blocks,
		threads,
		max_threads = prop.maxThreadsPerBlock,
		max_sm = prop.multiProcessorCount;

	blocks = matrix->core_count; //(matrix->var_count >= max_threads)? (matrix->var_count/max_threads)+1 : 1;
	/*
	if (matrix->thread_count < 0)
		threads = (matrix->var_count >= max_threads)? max_threads : matrix->var_count;
	else
		threads = (matrix->thread_count >= max_threads)? max_threads : matrix->thread_count;
	*/
	/*if (blocks > prop.multiProcessorCount)
		error_exit(3, (char*)"CUDA Device out of shared memory!");*/

	StartTimer();
	{
		naive_aproach_one_thread<<<blocks, threads>>>(matrix);
		checkCudaErrors(hipDeviceSynchronize());
	};

	//assert(threads*sizeof(double) <= prop.sharedMemPerBlock);
	double time = GetTimer()/1000;

	//check_solutions<<<1,threads, threads*sizeof(double)>>>(matrix->var_count, matrix->d_solution, matrix->d_reference);
	//checkCudaErrors(hipDeviceSynchronize());
	return time;
} 

__global__ void naive_aproach_one_thread(cuda_matrix *matrix){
	//missing!
}