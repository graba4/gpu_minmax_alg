#include "hip/hip_runtime.h"
#include "cuda_deque.h"
#include "implementations.h"
#include "io.h"
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "resources.h"
#include <timer.h>
#include "errors.h"
#include "cuda_schedule.h"

#define ROUND_UP(N, S) (N%S == 0) ? N/S : N/S+1
#define BETWEEN(value, min, max) (value < max && value > min)
#define DEV_ID (0)

__global__ void naive_aproach_one_thread(double *matrix, double *minval, double *maxval, int arrlen, int window_size);

__global__ void naive_aproach_multi_thread(double *matrix, double *minval, double *maxval, int arrlen, int window_size);

class Min_Max_calc
{
private:
	double min,
		   max;

public:
	__device__ Min_Max_calc(double *arr, unsigned int length){
		assert(arr != NULL);
		min = max = arr[0];
		for (int i = 1; i < length; ++i)
		{
			min = (arr[i] < min)? arr[i] : min;
			max = (arr[i] > max)? arr[i] : max;
		}
	}
	__device__ ~Min_Max_calc();

	__device__ double getMin(){
		return min;
	}

	__device__ double getMax(){
		return max;
	}
};

double naive_aproach_fabian(cuda_matrix *matrix){
	hipDeviceProp_t prop;
	checkCudaErrors(hipGetDeviceProperties(&prop, DEV_ID));

	int blocks,
		threads = matrix->thread_count,
		max_threads = prop.maxThreadsPerBlock,
		max_sm = prop.multiProcessorCount;

	blocks = matrix->core_count; //(matrix->var_count >= max_threads)? (matrix->var_count/max_threads)+1 : 1;
	/*
	if (matrix->thread_count < 0)
		threads = (matrix->var_count >= max_threads)? max_threads : matrix->var_count;
	else
		threads = (matrix->thread_count >= max_threads)? max_threads : matrix->thread_count;
	*/
	/*if (blocks > prop.multiProcessorCount)
		error_exit(3, (char*)"CUDA Device out of shared memory!");*/

	StartTimer();
	{
		naive_aproach_one_thread<<<blocks, threads>>>(matrix->d_matrix, matrix->d_minval, matrix->d_maxval, matrix->arrlen, matrix->window_size);
		checkCudaErrors(hipDeviceSynchronize());
	};

	//assert(threads*sizeof(double) <= prop.sharedMemPerBlock);
	double time = GetTimer()/1000;

	//check_solutions<<<1,threads, threads*sizeof(double)>>>(matrix->var_count, matrix->d_solution, matrix->d_reference);
	//checkCudaErrors(hipDeviceSynchronize());
	printf("Time: %f\n", time);
	return time;
}

double naive_aproach_amar(cuda_matrix *matrix){
	hipDeviceProp_t prop;
	checkCudaErrors(hipGetDeviceProperties(&prop, DEV_ID));

	int blocks,
		threads = matrix->thread_count,
		max_threads = prop.maxThreadsPerBlock,
		max_sm = prop.multiProcessorCount;

	blocks = matrix->core_count; //(matrix->var_count >= max_threads)? (matrix->var_count/max_threads)+1 : 1;
	/*
	if (matrix->thread_count < 0)
		threads = (matrix->var_count >= max_threads)? max_threads : matrix->var_count;
	else
		threads = (matrix->thread_count >= max_threads)? max_threads : matrix->thread_count;
	*/
	/*if (blocks > prop.multiProcessorCount)
		error_exit(3, (char*)"CUDA Device out of shared memory!");*/

	StartTimer();
	{
		naive_aproach_multi_thread<<<blocks, threads>>>(matrix->d_matrix, matrix->d_minval, matrix->d_maxval, matrix->arrlen, matrix->window_size);
		checkCudaErrors(hipDeviceSynchronize());
	};

	//assert(threads*sizeof(double) <= prop.sharedMemPerBlock);
	double time = GetTimer()/1000;

	//check_solutions<<<1,threads, threads*sizeof(double)>>>(matrix->var_count, matrix->d_solution, matrix->d_reference);
	//checkCudaErrors(hipDeviceSynchronize());
	printf("Time: %f\n", time);
	return time;
} 

__device__ void print_matrixx(double *matrix, int length)
{
	assert(matrix != NULL);

	/* image row */
	for (int i = 0; i < length; i++){
		printf("%.1f ", (matrix[i] == -0.0)? 0.0 : matrix[i]);
	}
	printf("\n");
}

#define MIN_WIN_SIZE (3)
__global__ void naive_aproach_one_thread(double *matrix, double *minval, double *maxval, int arrlen, int window_size){
	int tid = threadIdx.x,
		block_id = gridDim.x;
	assert(window_size >= MIN_WIN_SIZE);
	cuda_deque U, L;
	double *a = matrix;
	//print_matrixx(a, arrlen);

	if (tid == 0 && block_id == 1) //only one thread
	{
		for(u_int i = 1; i < arrlen; ++i){
			if(i >= window_size){
				maxval[i-window_size] = a[U.size() > 0 ? U.front():i-1];
				minval[i-window_size] = a[L.size() > 0 ? L.front():i-1];
			}

			if(a[i] > a[i-1]){
				L.push_back(i-1);
				
				if(i == window_size + L.front())
					L.pop_front();

				while(U.size() > 0){
					if(a[i] <= a[U.back()]){
						if(i == window_size + U.front())
							U.pop_front();
						break;
					}
					U.pop_back();
				}
			}else{
				U.push_back(i-1);
				
				if(i == window_size + U.front())
					U.pop_front();
				
				
				while(L.size() > 0){
					if(a[i] >= a[L.back()]){
						if(i == window_size + L.front())
							L.pop_front();
						break;
					}
					L.pop_back();
				}
			}
		}
		maxval[arrlen - window_size] = a[U.size() > 0 ? U.front() : arrlen-1];
		minval[arrlen - window_size] = a[L.size() > 0 ? L.front() : arrlen-1];
		
		//print_matrixx(maxval, arrlen);	
		//print_matrixx(minval, arrlen);
		
	}
}

__global__ void par_alg_inc_win(double *matrix, double *minval, double *maxval, int arrlen, int window_size){
	int tid = threadIdx.x,
		bid = blockIdx.x,
		shift_amount = 0;
	assert(window_size >= MIN_WIN_SIZE);

	int addr_offs = (tid + shift_amount) + bid*window_size*2;
	while(addr_offs+window_size < arrlen) {
		Min_Max_calc m((double *)(matrix + addr_offs));
		minval[addr_offs] = m.getMin();
		maxval[addr_offs] = m.getMax();

		addr_offs += gridDim.x;
	}
}

__global__ void par_alg_inc_blocks(double *matrix, double *minval, double *maxval, int arrlen, int window_size){
	int tid = threadIdx.x,
		bid = blockIdx.x,
		shift_amount = 0;
	assert(window_size >= MIN_WIN_SIZE);

	int addr_offs = (tid + shift_amount) + bid*window_size;
	while(addr_offs+window_size < arrlen) {
		Min_Max_calc m((double *)(matrix + addr_offs));
		minval[addr_offs] = m.getMin();
		maxval[addr_offs] = m.getMax();

		addr_offs += window_size;
	}
}

__global__ void naive_aproach_multi_thread(double *matrix, double *minval, double *maxval, int arrlen, int window_size){
	int tid = threadIdx.x,
		block_id = gridDim.x;
	cuda_deque U, L;
	double *a = matrix;
	//print_matrixx(a, arrlen);
	//printf("%d %d\n", tid, block_id);
	if(tid == 0){
		for(u_int i = 1; i < arrlen; ++i){
			if(i >= window_size){
				maxval[i-window_size] = a[U.size() > 0 ? U.front():i-1];
			}				

			if(a[i] > a[i-1]){
				while(U.size() > 0){
					if(a[i] <= a[U.back()]){
						if(i == window_size + U.front())
							U.pop_front();
						break;
					}
					U.pop_back();
				}
			}else{
				U.push_back(i-1);				
				if(i == window_size + U.front())
					U.pop_front();
			}
		}
		maxval[arrlen - window_size] = a[U.size() > 0 ? U.front() : arrlen-1];
		//printf("%d %d\n", tid, block_id);	
	}
	if(tid == 1){
		//printf("%d %d\n", tid, block_id);	
		for(u_int i = 1; i < arrlen; ++i){
			if(i >= window_size){
				minval[i-window_size] = a[L.size() > 0 ? L.front():i-1];
			}
			if(a[i] > a[i-1]){
				L.push_back(i-1);
				
				if(i == window_size + L.front())
					L.pop_front();

			}else{
				while(L.size() > 0){
					if(a[i] >= a[L.back()]){
						if(i == window_size + L.front())
							L.pop_front();
						break;
					}
					L.pop_back();
				}
			}
		}
		minval[arrlen - window_size] = a[L.size() > 0 ? L.front() : arrlen-1];		
		//printf("%d %d\n", tid, block_id);
	}
	//print_matrixx(maxval, arrlen);	
	//print_matrixx(minval, arrlen);
}